#include "hip/hip_runtime.h"
#include "kernel.cuh"

//convert the pixel into eight bits
__global__ void bitExtractation(uchar3 * deviceFrame, uchar3 * deviceTempBitMatrix, int frameWidth){
    //calculate the thread index
    int threadIndexX = threadIdx.y + blockIdx.y * blockDim.y;
    int threadIndexY = threadIdx.x + blockIdx.x * blockDim.x;
    int threadIndexZ = threadIdx.z;

    //blue channel
    if(threadIndexZ == 0){
        unsigned char pixelValue = deviceFrame[threadIndexY * frameWidth + threadIndexX].x;

        for(int i = 0; i < 8; i ++){
            unsigned char temp                                                        = pixelValue;
            deviceTempBitMatrix[(threadIndexY * frameWidth + threadIndexX) * 8 + i].x = temp & 0x01;
            pixelValue                                                                = pixelValue >> 1;
        }
    }

    //green channel
    else if(threadIndexZ == 1){
        unsigned char pixelValue = deviceFrame[threadIndexY * frameWidth + threadIndexX].y;

        for(int i = 0; i < 8; i ++){
            unsigned char temp                                                        = pixelValue;
            deviceTempBitMatrix[(threadIndexY * frameWidth + threadIndexX) * 8 + i].y = temp & 0x01;
            pixelValue                                                                = pixelValue >> 1;
        }
    }

    //red channel
    else{
        unsigned char pixelValue = deviceFrame[threadIndexY * frameWidth + threadIndexX].z;

        for(int i = 0; i < 8; i ++){
            unsigned char temp                                                        = pixelValue;
            deviceTempBitMatrix[(threadIndexY * frameWidth + threadIndexX) * 8 + i].z = temp & 0x01;
            pixelValue                                                                = pixelValue >> 1;
        }
    }
}

//reconstruct pixel using bits
__global__ void pixelReconstruction(uchar3 * deviceTempBitMatrix, uchar3 * deviceFrame, int frameWidth){
    //calculate the thread index
    int threadIndexX = threadIdx.y + blockIdx.y * blockDim.y;
    int threadIndexY = threadIdx.x + blockIdx.x * blockDim.x;
    int threadIndexZ = threadIdx.z;

    unsigned char pixelValue = 0;

    //blue channel
    if(threadIndexZ == 0){
        for(int i = 7; i >=0; i --){
            pixelValue = pixelValue | deviceTempBitMatrix[(threadIndexY * frameWidth + threadIndexX) * 8 + i].x;
            if(i != 0)
                pixelValue = pixelValue << 1;
        }

        deviceFrame[threadIndexY * frameWidth + threadIndexX].x = pixelValue;
    }

    //green channel
    else if(threadIndexZ == 1){
        for(int i = 7; i >=0; i --){
            pixelValue = pixelValue | deviceTempBitMatrix[(threadIndexY * frameWidth + threadIndexX) * 8 + i].y;
            if(i != 0)
                pixelValue = pixelValue << 1;
        }

        deviceFrame[threadIndexY * frameWidth + threadIndexX].y = pixelValue;
    }

    //red channel
    else{
        for(int i = 7; i >=0; i --){
            pixelValue = pixelValue | deviceTempBitMatrix[(threadIndexY * frameWidth + threadIndexX) * 8 + i].z;
            if(i != 0)
                pixelValue = pixelValue << 1;
        }

        deviceFrame[threadIndexY * frameWidth + threadIndexX].z = pixelValue;
    }
}

//perform confusion operations along horizontal direction
__global__ void confusionAlongHorizontalDirection(uchar3 * deviceTempBitMatrix, uchar3 * deviceEncryptedBitMatrix, 
                                                  uint16_t * deviceShiftDistanceSequence, int frameWidth){
    //calculate the thread index
    int threadIndexX = threadIdx.y + blockIdx.y * blockDim.y;
    int threadIndexY = threadIdx.x + blockIdx.x * blockDim.x;
    int threadIndexZ = threadIdx.z;

    //blue channel
    if(threadIndexZ == 0){
        uint16_t shiftDistance = deviceShiftDistanceSequence[(threadIndexY * 3)] % frameWidth;
        int newThreadIndexX    = (threadIndexX + shiftDistance) % frameWidth;

        for(int i = 0; i < 8; i++)
            deviceEncryptedBitMatrix[(threadIndexY * frameWidth + newThreadIndexX) * 8 + i].x = deviceTempBitMatrix[(threadIndexY * frameWidth + threadIndexX) * 8 + i].x;
    }

    //green channel
    else if(threadIndexZ == 1){
        uint16_t shiftDistance = deviceShiftDistanceSequence[(threadIndexY * 3) + 1] % frameWidth;
        int newThreadIndexX    = (threadIndexX + shiftDistance) % frameWidth;

        for(int i = 0; i < 8; i++)
            deviceEncryptedBitMatrix[(threadIndexY * frameWidth + newThreadIndexX) * 8 + i].y = deviceTempBitMatrix[(threadIndexY * frameWidth + threadIndexX) * 8 + i].y;
    }

    //red channel
    else{
        uint16_t shiftDistance = deviceShiftDistanceSequence[(threadIndexY * 3) + 2] % frameWidth;
        int newThreadIndexX    = (threadIndexX + shiftDistance) % frameWidth;

        for(int i = 0; i < 8; i++)
            deviceEncryptedBitMatrix[(threadIndexY * frameWidth + newThreadIndexX) * 8 + i].z = deviceTempBitMatrix[(threadIndexY * frameWidth + threadIndexX) * 8 + i].z;
    }
}

//perform confusion operations along vertical direction
__global__ void confusionAlongVerticalDirection(uchar3 * deviceTempBitMatrix, uchar3 * deviceEncryptedBitMatrix,
                                                uint16_t * deviceShiftDistanceSequence, int frameWidth, int frameHeight){
    //calculate the thread index
    int threadIndexX = threadIdx.y + blockIdx.y * blockDim.y;
    int threadIndexY = threadIdx.x + blockIdx.x * blockDim.x;
    int threadIndexZ = threadIdx.z;

    //blue channel
    if(threadIndexZ == 0){
        uint16_t shiftDistance = deviceShiftDistanceSequence[frameHeight * 3 + (threadIndexX * 3)] % frameHeight;
        int newThreadIndexY    = (threadIndexY + shiftDistance) % frameHeight;

        for(int i = 0; i < 8; i ++)
            deviceEncryptedBitMatrix[(newThreadIndexY * frameWidth + threadIndexX) * 8 + i].x = deviceTempBitMatrix[(threadIndexY * frameWidth + threadIndexX) * 8 + i].x;
    }

    //green channel
    else if(threadIndexZ == 1){
        uint16_t shiftDistance = deviceShiftDistanceSequence[frameHeight * 3 + (threadIndexX * 3) + 1] % frameHeight;
        int newThreadIndexY    = (threadIndexY + shiftDistance) % frameHeight;

        for(int i = 0; i < 8; i ++)
            deviceEncryptedBitMatrix[(newThreadIndexY * frameWidth + threadIndexX) * 8 + i].y = deviceTempBitMatrix[(threadIndexY * frameWidth + threadIndexX) * 8 + i].y;
    }

    //red channel
    else{
        uint16_t shiftDistance = deviceShiftDistanceSequence[frameHeight * 3 + (threadIndexX * 3) + 2] % frameHeight;
        int newThreadIndexY    = (threadIndexY + shiftDistance) % frameHeight;

        for(int i = 0; i < 8; i ++)
            deviceEncryptedBitMatrix[(newThreadIndexY * frameWidth + threadIndexX) * 8 + i].z = deviceTempBitMatrix[(threadIndexY * frameWidth + threadIndexX) * 8 + i].z;
    }
}

//perform XOR operation to encrypt or decrypt the pixel
__global__ void XOROperations(uchar3 * deviceTempBitMatrix, uchar3 * deviceResultBitMatrix, unsigned char * deviceByteSequence, int frameWidth){
    //calculate the thread index
    int threadIndexX = threadIdx.y + blockIdx.y * blockDim.y;
    int threadIndexY = threadIdx.x + blockIdx.x * blockDim.x;
    int threadIndexZ = threadIdx.z;

    if(threadIndexZ == 0){
        unsigned char byteValue = deviceByteSequence[(threadIndexY * frameWidth) * 3 + threadIndexX * 3];

        for(int i = 0; i < 8; i ++){
            unsigned char temp = 0;
            temp               = byteValue & 0x01;
            deviceResultBitMatrix[(threadIndexY * frameWidth + threadIndexX) * 8 + i].x = deviceTempBitMatrix[(threadIndexY * frameWidth + threadIndexX) * 8 + i].x ^ temp;
            byteValue          = byteValue >> 1;
        }
    }

    else if(threadIndexZ == 1){
        unsigned char byteValue = deviceByteSequence[(threadIndexY * frameWidth) * 3 + threadIndexX * 3 + 1];

        for(int i = 0; i < 8; i ++){
            unsigned char temp = 0;
            temp               = byteValue & 0x01;
            deviceResultBitMatrix[(threadIndexY * frameWidth + threadIndexX) * 8 + i].y = deviceTempBitMatrix[(threadIndexY * frameWidth + threadIndexX) * 8 + i].y ^ temp;
            byteValue          = byteValue >> 1;
        }
    }

    else{
        unsigned char byteValue = deviceByteSequence[(threadIndexY * frameWidth) * 3 + threadIndexX * 3 + 2];

        for(int i = 0; i < 8; i ++){
            unsigned char temp = 0;
            temp               = byteValue & 0x01;
            deviceResultBitMatrix[(threadIndexY * frameWidth + threadIndexX) * 8 + i].z = deviceTempBitMatrix[(threadIndexY * frameWidth + threadIndexX) * 8 + i].z ^ temp;
            byteValue          = byteValue >> 1;
        }
    }
}

//the main thread call this function to encrypt the original frame using GPU
extern "C"
void encryptionKernelCaller(uchar3 * deviceOriginalFrame, uchar3 * deviceEncryptedFrame, uchar3 * deviceEncryptedBitMatrix, uchar3 * deviceTempBitMatrix,
                            uint16_t * deviceShiftDistanceSequence, unsigned char * deviceByteSequence, int frameWidth, int frameHeight, int frameDataLength){

    //Each block consists of (1, GPU_BLOCK_LENGTH, 3) GPU threads, with a total of (frameHeight, frameWidth / GPU_BLOCK_LENGTH) blocks.
    dim3 block(1, GPU_BLOCK_LENGTH, 3);
    dim3 grid(frameHeight, frameWidth / GPU_BLOCK_LENGTH);

    //convert the original frame into bit matrix
    bitExtractation<<<grid, block>>>(deviceOriginalFrame, deviceTempBitMatrix, frameWidth);
    hipDeviceSynchronize();

    //perform confusion operations along horizontal direction
    confusionAlongHorizontalDirection<<<grid, block>>>(deviceTempBitMatrix, deviceEncryptedBitMatrix, deviceShiftDistanceSequence, frameWidth);
    hipDeviceSynchronize();
    hipMemcpy(deviceTempBitMatrix, deviceEncryptedBitMatrix, frameDataLength * 8, hipMemcpyDeviceToDevice);

    //perform confusion operations along vertical direction
    confusionAlongVerticalDirection<<<grid, block>>>(deviceTempBitMatrix, deviceEncryptedBitMatrix, deviceShiftDistanceSequence, frameWidth, frameHeight);
    hipDeviceSynchronize();
    hipMemcpy(deviceTempBitMatrix, deviceEncryptedBitMatrix, frameDataLength * 8, hipMemcpyDeviceToDevice);

    //perform XOR operations to encrypt the shuffled bit matrix
    XOROperations<<<grid, block>>>(deviceTempBitMatrix, deviceEncryptedBitMatrix, deviceByteSequence, frameWidth);
    hipDeviceSynchronize();
    hipMemcpy(deviceTempBitMatrix, deviceEncryptedBitMatrix, frameDataLength * 8, hipMemcpyDeviceToDevice);

    //convert the encrypted frame into pixel level frame
    pixelReconstruction<<<grid, block>>>(deviceTempBitMatrix, deviceEncryptedFrame, frameWidth);
    hipDeviceSynchronize();
}
